#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2015 by Contributors
 * \file upsampling_nearest.cc
 * \brief
 * \author Bing Xu
*/

#include "./deconvolution-inl.h"
#include "./upsampling-inl.h"

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(UpSamplingParam param) {
  if (param.sample_type == up_enum::kNearest) {
    return new UpSamplingNearestOp<gpu>(param);
  } else if (param.sample_type == up_enum::kBilinear) {
    DeconvolutionParam p;
    int kernel = 2 * param.scale - param.scale % 2;
    int stride = param.scale;
    int pad = static_cast<int>(ceil((param.scale - 1) / 2.));
    p.num_group = param.num_filter;
    p.num_filter = param.num_filter;
    p.no_bias =  true;
    int shape[] = {1, 1};
    shape[0] = shape[1] = kernel;
    p.kernel = TShape(shape, shape + 2);
    shape[0] = shape[1] = stride;
    p.stride = TShape(shape, shape + 2);
    shape[0] = shape[1] = pad;
    p.pad = TShape(shape, shape + 2);
    return new DeconvolutionOp<gpu>(p);
  } else {
    LOG(FATAL) << "Unknown sample type";
    return NULL;
  }
}

}  // namespace op
}  // namespace mxnet
